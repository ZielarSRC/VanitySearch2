// GPUEngine.cu
#include "GPUEngine.h"
#include "GPUKernels.cuh"
#include <stdexcept>
#include <cstdio>
#include <cstring>

#define _64K 65536
#define ITEM_SIZE 40
#define ITEM_SIZE32 (ITEM_SIZE / 4)
#define GRP_SIZE 1024
#define STEP_SIZE 256
#define BIFULLSIZE 32

namespace GPUEngine {

namespace {
    constexpr const char* CUDA_ERROR_PREFIX = "CUDA error: ";

    void checkCudaError(hipError_t err, const char* context) {
        if (err != hipSuccess) {
            throw std::runtime_error(std::string(CUDA_ERROR_PREFIX) + context + ": " + hipGetErrorString(err));
        }
    }

    int convertSMVerToCores(int major, int minor) {
        struct SMToCores { int version; int cores; };
        const SMToCores archCoresPerSM[] = {
            {0x20, 32}, {0x21, 48}, {0x30, 192}, {0x32, 192},
            {0x35, 192}, {0x37, 192}, {0x50, 128}, {0x52, 128},
            {0x53, 128}, {0x60, 64}, {0x61, 128}, {0x62, 128},
            {0x70, 64}, {0x72, 64}, {0x75, 64}, {0x80, 64},
            {0x86, 128}, {-1, -1}
        };

        const int version = (major << 4) + minor;
        for (const auto& entry : archCoresPerSM) {
            if (entry.version == version) return entry.cores;
        }
        return 0;
    }
}

CudaContext::CudaContext(int device_id) : device_id_(device_id) {
    checkCudaError(hipSetDevice(device_id_), "Set device");
    checkCudaError(hipGetDeviceProperties(&prop_, device_id_), "Get device properties");
    checkCudaError(hipDeviceSetCacheConfig(hipFuncCachePreferL1), "Set cache config");
    checkCudaError(hipDeviceSetLimit(hipLimitStackSize, 49152), "Set stack size");
}

CudaContext::~CudaContext() {
    if (device_id_ >= 0) hipDeviceReset();
}

GPUMemory::GPUMemory(size_t size) { allocate(size); }
GPUMemory::~GPUMemory() { free(); }

void GPUMemory::allocate(size_t size) {
    if (ptr_) free();
    checkCudaError(hipMalloc(&ptr_, size), "Allocate device memory");
    size_ = size;
}

void GPUMemory::free() {
    if (ptr_) { hipFree(ptr_); ptr_ = nullptr; size_ = 0; }
}

PinnedMemory::PinnedMemory(size_t size, unsigned int flags) { allocate(size, flags); }
PinnedMemory::~PinnedMemory() { free(); }

void PinnedMemory::allocate(size_t size, unsigned int flags) {
    if (ptr_) free();
    checkCudaError(hipHostAlloc(&ptr_, size, flags), "Allocate pinned memory");
    size_ = size;
}

void PinnedMemory::free() {
    if (ptr_) { hipHostFree(ptr_); ptr_ = nullptr; size_ = 0; }
}

Engine::Engine(int threadGroups, int threadsPerGroup, int gpuId, uint32_t maxFound, bool rekey) 
    : totalThreads_(0), threadsPerGroup_(threadsPerGroup), maxFound_(maxFound), 
      rekeyEnabled_(rekey), searchMode_(SearchMode::Compressed), searchType_(SearchType::P2PKH) {
    try {
        context_ = std::make_unique<CudaContext>(gpuId);
        initialize();
    } catch (...) {
        cleanup();
        throw;
    }
}

Engine::~Engine() { cleanup(); }

void Engine::initialize() {
    const auto& props = context_->properties();
    if (threadGroups == -1) threadGroups = props.multiProcessorCount * 8;
    
    totalThreads_ = threadGroups * threadsPerGroup_;
    outputSize_ = (maxFound_ * ITEM_SIZE + 4);

    char buffer[512];
    snprintf(buffer, sizeof(buffer), "GPU #%d %s (%dx%d cores) Grid(%dx%d)",
            context_->device_id(), props.name, props.multiProcessorCount,
            convertSMVerToCores(props.major, props.minor),
            totalThreads_ / threadsPerGroup_, threadsPerGroup_);
    deviceName_ = buffer;

    devicePrefixes_.allocate(_64K * 2);
    deviceKeys_.allocate(totalThreads_ * 32 * 2);
    deviceOutput_.allocate(outputSize_);
    hostPrefixes_.allocate(_64K * 2, hipHostMallocWriteCombined | hipHostMallocMapped);
    hostKeys_.allocate(totalThreads_ * 32 * 2, hipHostMallocWriteCombined | hipHostMallocMapped);
    hostOutput_.allocate(outputSize_, hipHostMallocMapped);

    initialized_ = true;
}

// Kontynuacja GPUEngine.cu
void Engine::cleanup() {
    if (computeStream_) hipStreamDestroy(computeStream_);
    if (memcpyStream_) hipStreamDestroy(memcpyStream_);
    if (computeDoneEvent_) hipEventDestroy(computeDoneEvent_);
}

void Engine::SetPrefixes(const std::vector<uint16_t>& prefixes) {
    if (!initialized_) throw std::runtime_error("Engine not initialized");
    
    std::memset(hostPrefixes_.get(), 0, _64K * 2);
    auto hostPtr = static_cast<uint16_t*>(hostPrefixes_.get());
    for (auto prefix : prefixes) hostPtr[prefix] = 1;
    
    checkCudaError(hipMemcpy(devicePrefixes_.get(), hostPrefixes_.get(), 
                  _64K * 2, hipMemcpyHostToDevice), "Copy prefixes");
    
    if (!rekeyEnabled_) hostPrefixes_.free();
}

bool Engine::SetKeys(const std::vector<Point>& points) {
    if (!initialized_) return false;
    if (points.size() < static_cast<size_t>(totalThreads_)) {
        throw std::runtime_error("Insufficient points for threads");
    }

    auto hostPtr = static_cast<uint64_t*>(hostKeys_.get());
    for (int i = 0; i < totalThreads_; i++) {
        const auto& point = points[i];
        for (int j = 0; j < 4; j++) {
            hostPtr[i * 8 + j] = point.x[j];
            hostPtr[i * 8 + j + 4] = point.y[j];
        }
    }

    checkCudaError(hipMemcpy(deviceKeys_.get(), hostKeys_.get(), 
                  totalThreads_ * 32 * 2, hipMemcpyHostToDevice), "Copy keys");

    if (!rekeyEnabled_) hostKeys_.free();
    return callKernel();
}

bool Engine::callKernel() {
    if (!initialized_) return false;

    dim3 grid(totalThreads_ / threadsPerGroup_);
    dim3 block(threadsPerGroup_);

    void* kernelFunc = nullptr;
    if (searchType_ == SearchType::P2SH) {
        kernelFunc = hasPattern_ ? (void*)comp_keys_p2sh_pattern : (void*)comp_keys_p2sh;
    } else {
        kernelFunc = hasPattern_ ? (void*)comp_keys_pattern : 
                   (searchMode_ == SearchMode::Compressed) ? (void*)comp_keys_comp : (void*)comp_keys;
    }

    void* args[] = {
        &searchMode_, &devicePrefixes_.get(), &devicePrefixLookup_.get(), 
        &deviceKeys_.get(), &maxFound_, &deviceOutput_.get()
    };

    checkCudaError(hipLaunchKernel(reinterpret_cast<const void*>(kernelFunc), grid, block, args, 0, computeStream_), "Launch kernel");
    return true;
}

bool Engine::Launch(std::vector<Item>& foundItems, bool spinWait) {
    if (!initialized_) return false;
    foundItems.clear();

    if (spinWait) {
        context_->synchronize();
        checkCudaError(hipMemcpy(hostOutput_.get(), deviceOutput_.get(), 
                      outputSize_, hipMemcpyDeviceToHost), "Copy output");
    } else {
        checkCudaError(hipMemcpyAsync(hostOutput_.get(), deviceOutput_.get(), 
                      4, hipMemcpyDeviceToHost, memcpyStream_), "Async copy");
        checkCudaError(hipStreamSynchronize(memcpyStream_), "Sync stream");
    }

    auto hostPtr = static_cast<uint32_t*>(hostOutput_.get());
    uint32_t foundCount = hostPtr[0];
    if (foundCount > maxFound_) foundCount = maxFound_;

    if (foundCount > 0) {
        checkCudaError(hipMemcpy(hostOutput_.get(), deviceOutput_.get(), 
                      foundCount * ITEM_SIZE + 4, hipMemcpyDeviceToHost), "Copy results");

        foundItems.reserve(foundCount);
        for (uint32_t i = 0; i < foundCount; i++) {
            uint32_t* itemPtr = hostPtr + (i * ITEM_SIZE32 + 1);
            Item item;
            item.threadId = itemPtr[0];
            item.endomorphism = reinterpret_cast<int16_t*>(&itemPtr[1])[0] & 0x7FFF;
            item.mode = (reinterpret_cast<int16_t*>(&itemPtr[1])[0] & 0x8000) != 0;
            item.increment = reinterpret_cast<int16_t*>(&itemPtr[1])[1];
            item.hash = reinterpret_cast<uint8_t*>(itemPtr + 2);
            foundItems.push_back(item);
        }
    }
    return callKernel();
}

void Engine::PrintCudaInfo() {
    int deviceCount = 0;
    if (hipGetDeviceCount(&deviceCount) != hipSuccess || deviceCount == 0) {
        printf("No CUDA devices found\n");
        return;
    }

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("GPU #%d %s (%dx%d cores) (%.1f MB)\n",
               i, props.name, props.multiProcessorCount,
               convertSMVerToCores(props.major, props.minor),
               props.totalGlobalMem / 1048576.0);
    }
}
