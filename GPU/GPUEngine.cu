#include "GPUEngine.h"
#include <hip/hip_runtime.h>
#include <memory>
#include <stdexcept>
#include "../Timer.h"
#include "GPUGroup.h"
#include "GPUMath.h"
#include "GPUHash.h"
#include "GPUBase58.h"
#include "GPUWildcard.h"
#include "GPUCompute.h"

namespace GPUEngine {

    namespace {
        constexpr const char* CUDA_ERROR_PREFIX = "CUDA error: ";

        void CheckCudaError(hipError_t err, const char* context) {
            if (err != hipSuccess) {
                throw std::runtime_error(std::string(CUDA_ERROR_PREFIX) + 
                       context + ": " + hipGetErrorString(err));
            }
        }

        int ConvertSMVerToCores(int major, int minor) {
            struct SMToCores {
                int version;
                int cores;
            };

            const SMToCores archCoresPerSM[] = {
                {0x20, 32}, {0x21, 48}, {0x30, 192}, {0x32, 192},
                {0x35, 192}, {0x37, 192}, {0x50, 128}, {0x52, 128},
                {0x53, 128}, {0x60, 64}, {0x61, 128}, {0x62, 128},
                {0x70, 64}, {0x72, 64}, {0x75, 64}, {0x80, 64},
                {0x86, 128}, {-1, -1}
            };

            const int version = (major << 4) + minor;
            for (const auto& entry : archCoresPerSM) {
                if (entry.version == version) {
                    return entry.cores;
                }
            }
            return 0;
        }
    }

    Engine::Engine(int threadGroups, int threadsPerGroup, int gpuId, 
                  uint32_t maxFound, bool rekey) 
        : totalThreads_(0), threadsPerGroup_(threadsPerGroup),
          initialized_(false), rekeyEnabled_(rekey), maxFound_(maxFound),
          hasPattern_(false) {
        
        try {
            int deviceCount = 0;
            CheckCudaError(hipGetDeviceCount(&deviceCount), "Get device count");

            if (deviceCount == 0) {
                throw std::runtime_error("No CUDA-capable devices available");
            }

            CheckCudaError(hipSetDevice(gpuId), "Set device");

            hipDeviceProp_t props;
            CheckCudaError(hipGetDeviceProperties(&props, gpuId), "Get device properties");

            if (threadGroups == -1) {
                threadGroups = props.multiProcessorCount * 8;
            }

            totalThreads_ = threadGroups * threadsPerGroup;
            outputSize_ = (maxFound * ITEM_SIZE + 4);

            char buffer[512];
            snprintf(buffer, sizeof(buffer),
                    "GPU #%d %s (%dx%d cores) Grid(%dx%d)",
                    gpuId, props.name, props.multiProcessorCount,
                    ConvertSMVerToCores(props.major, props.minor),
                    totalThreads_ / threadsPerGroup, threadsPerGroup);
            deviceName_ = buffer;

            // Configure device
            CheckCudaError(hipDeviceSetCacheConfig(hipFuncCachePreferL1), "Set cache config");
            CheckCudaError(hipDeviceSetLimit(hipLimitStackSize, 49152), "Set stack size");

            // Allocate device memory
            CheckCudaError(hipMalloc(&devicePrefixes_, _64K * 2), "Allocate prefixes");
            CheckCudaError(hipHostAlloc(&hostPrefixes_, _64K * 2, 
                          hipHostMallocWriteCombined | hipHostMallocMapped), 
                          "Allocate pinned prefixes");
            
            CheckCudaError(hipMalloc(&deviceKeys_, totalThreads_ * 32 * 2), "Allocate keys");
            CheckCudaError(hipHostAlloc(&hostKeys_, totalThreads_ * 32 * 2,
                          hipHostMallocWriteCombined | hipHostMallocMapped),
                          "Allocate pinned keys");

            CheckCudaError(hipMalloc(&deviceOutput_, outputSize_), "Allocate output");
            CheckCudaError(hipHostAlloc(&hostOutput_, outputSize_, hipHostMallocMapped),
                          "Allocate pinned output");

            searchMode_ = SearchMode::Compressed;
            searchType_ = SearchType::P2PKH;
            initialized_ = true;

        } catch (const std::exception& e) {
            Cleanup();
            throw;
        }
    }

    void Engine::Cleanup() {
        if (deviceKeys_) hipFree(deviceKeys_);
        if (devicePrefixes_) hipFree(devicePrefixes_);
        if (devicePrefixLookup_) hipFree(devicePrefixLookup_);
        if (hostOutput_) hipHostFree(hostOutput_);
        if (deviceOutput_) hipFree(deviceOutput_);
        if (hostPrefixes_) hipHostFree(hostPrefixes_);
        if (hostKeys_) hipHostFree(hostKeys_);
        if (hostPrefixLookup_) hipHostFree(hostPrefixLookup_);
    }

    Engine::~Engine() {
        Cleanup();
    }

        void Engine::SetPrefixes(const std::vector<PrefixT>& prefixes) {
        if (!initialized_) {
            throw std::runtime_error("Engine not initialized");
        }

        std::memset(hostPrefixes_, 0, _64K * 2);
        for (const auto prefix : prefixes) {
            hostPrefixes_[prefix] = 1;
        }

        CheckCudaError(hipMemcpy(devicePrefixes_, hostPrefixes_, _64K * 2, hipMemcpyHostToDevice),
                      "Copy prefixes to device");

        if (!rekeyEnabled_) {
            hipHostFree(hostPrefixes_);
            hostPrefixes_ = nullptr;
        }

        lostWarning_ = false;
    }

    void Engine::SetPrefixes(const std::vector<LPrefix>& prefixes, uint32_t totalPrefix) {
        if (!initialized_) {
            throw std::runtime_error("Engine not initialized");
        }

        // Allocate lookup memory
        CheckCudaError(hipMalloc(&devicePrefixLookup_, (_64K + totalPrefix) * 4),
                      "Allocate prefix lookup");
        CheckCudaError(hipHostAlloc(&hostPrefixLookup_, (_64K + totalPrefix) * 4,
                      hipHostMallocWriteCombined | hipHostMallocMapped),
                      "Allocate pinned prefix lookup");

        std::memset(hostPrefixes_, 0, _64K * 2);
        std::memset(hostPrefixLookup_, 0, _64K * 4);

        uint32_t offset = _64K;
        for (const auto& prefix : prefixes) {
            int count = static_cast<int>(prefix.longPrefixes.size());
            hostPrefixes_[prefix.shortPrefix] = static_cast<PrefixT>(count);
            hostPrefixLookup_[prefix.shortPrefix] = offset;
            
            for (const auto longPrefix : prefix.longPrefixes) {
                hostPrefixLookup_[offset++] = longPrefix;
            }
        }

        if (offset != (_64K + totalPrefix)) {
            throw std::runtime_error("Mismatch in total prefix count");
        }

        // Copy to device
        CheckCudaError(hipMemcpy(devicePrefixes_, hostPrefixes_, _64K * 2, hipMemcpyHostToDevice),
                      "Copy prefixes to device");
        CheckCudaError(hipMemcpy(devicePrefixLookup_, hostPrefixLookup_, 
                      (_64K + totalPrefix) * 4, hipMemcpyHostToDevice),
                      "Copy prefix lookup to device");

        // Free host memory
        hipHostFree(hostPrefixes_);
        hostPrefixes_ = nullptr;
        hipHostFree(hostPrefixLookup_);
        hostPrefixLookup_ = nullptr;
        
        lostWarning_ = false;
    }

    void Engine::SetPattern(const std::string& pattern) {
        if (!initialized_) {
            throw std::runtime_error("Engine not initialized");
        }

        if (pattern.size() >= _64K * 2) {
            throw std::runtime_error("Pattern too large");
        }

        std::memcpy(hostPrefixes_, pattern.data(), pattern.size());
        CheckCudaError(hipMemcpy(devicePrefixes_, hostPrefixes_, _64K * 2, hipMemcpyHostToDevice),
                      "Copy pattern to device");

        hipHostFree(hostPrefixes_);
        hostPrefixes_ = nullptr;
        
        lostWarning_ = false;
        hasPattern_ = true;
        searchPattern_ = pattern;
    }

    bool Engine::SetKeys(const std::vector<Point>& points) {
        if (!initialized_) {
            return false;
        }

        if (points.size() < static_cast<size_t>(totalThreads_)) {
            throw std::runtime_error("Insufficient points for threads");
        }

        for (int i = 0; i < totalThreads_; i += threadsPerGroup_) {
            for (int j = 0; j < threadsPerGroup_; j++) {
                const auto& point = points[i + j];
                const int baseIdx = 8 * i + j;

                hostKeys_[baseIdx + 0 * threadsPerGroup_] = point.x.bits64[0];
                hostKeys_[baseIdx + 1 * threadsPerGroup_] = point.x.bits64[1];
                hostKeys_[baseIdx + 2 * threadsPerGroup_] = point.x.bits64[2];
                hostKeys_[baseIdx + 3 * threadsPerGroup_] = point.x.bits64[3];
                hostKeys_[baseIdx + 4 * threadsPerGroup_] = point.y.bits64[0];
                hostKeys_[baseIdx + 5 * threadsPerGroup_] = point.y.bits64[1];
                hostKeys_[baseIdx + 6 * threadsPerGroup_] = point.y.bits64[2];
                hostKeys_[baseIdx + 7 * threadsPerGroup_] = point.y.bits64[3];
            }
        }

        CheckCudaError(hipMemcpy(deviceKeys_, hostKeys_, totalThreads_ * 32 * 2, hipMemcpyHostToDevice),
                      "Copy keys to device");

        if (!rekeyEnabled_) {
            hipHostFree(hostKeys_);
            hostKeys_ = nullptr;
        }

        return CallKernel();
    }

    bool Engine::CallKernel() {
        if (!initialized_) {
            return false;
        }

        // Reset found count
        CheckCudaError(hipMemset(deviceOutput_, 0, 4), "Reset output buffer");

        const dim3 grid(totalThreads_ / threadsPerGroup_);
        const dim3 block(threadsPerGroup_);

        if (searchType_ == SearchType::P2SH) {
            if (hasPattern_) {
                comp_keys_p2sh_pattern<<<grid, block>>>(
                    static_cast<uint32_t>(searchMode_),
                    devicePrefixes_,
                    deviceKeys_,
                    maxFound_,
                    deviceOutput_);
            } else {
                comp_keys_p2sh<<<grid, block>>>(
                    static_cast<uint32_t>(searchMode_),
                    devicePrefixes_,
                    devicePrefixLookup_,
                    deviceKeys_,
                    maxFound_,
                    deviceOutput_);
            }
        } else {
            if (hasPattern_) {
                if (searchType_ == SearchType::BECH32) {
                    throw std::runtime_error("BECH32 not yet supported with wildcard");
                }
                comp_keys_pattern<<<grid, block>>>(
                    static_cast<uint32_t>(searchMode_),
                    devicePrefixes_,
                    deviceKeys_,
                    maxFound_,
                    deviceOutput_);
            } else {
                if (searchMode_ == SearchMode::Compressed) {
                    comp_keys_comp<<<grid, block>>>(
                        devicePrefixes_,
                        devicePrefixLookup_,
                        deviceKeys_,
                        maxFound_,
                        deviceOutput_);
                } else {
                    comp_keys<<<grid, block>>>(
                        static_cast<uint32_t>(searchMode_),
                        devicePrefixes_,
                        devicePrefixLookup_,
                        deviceKeys_,
                        maxFound_,
                        deviceOutput_);
                }
            }
        }

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error(std::string("Kernel launch failed: ") + hipGetErrorString(err));
        }

        return true;
    }

    bool Engine::Launch(std::vector<Item>& foundItems, bool spinWait) {
        if (!initialized_) {
            return false;
        }

        foundItems.clear();

        if (spinWait) {
            CheckCudaError(hipMemcpy(hostOutput_, deviceOutput_, outputSize_, hipMemcpyDeviceToHost),
                          "Copy output (spin wait)");
        } else {
            hipEvent_t event;
            CheckCudaError(hipEventCreate(&event), "Create event");
            CheckCudaError(hipMemcpyAsync(hostOutput_, deviceOutput_, 4, hipMemcpyDeviceToHost, 0),
                          "Async copy output");
            CheckCudaError(hipEventRecord(event, 0), "Record event");

            while (hipEventQuery(event) == hipErrorNotReady) {
                Timer::SleepMillis(1);
            }
            CheckCudaError(hipEventDestroy(event), "Destroy event");
        }

        uint32_t foundCount = hostOutput_[0];
        if (foundCount > maxFound_) {
            if (!lostWarning_) {
                printf("\nWarning, %d items lost\nHint: Search with less prefixes, less threads (-g) or increase maxFound (-m)\n",
                      (foundCount - maxFound_));
                lostWarning_ = true;
            }
            foundCount = maxFound_;
        }

        if (foundCount > 0) {
            CheckCudaError(hipMemcpy(hostOutput_, deviceOutput_, foundCount * ITEM_SIZE + 4, hipMemcpyDeviceToHost),
                          "Copy found items");

            foundItems.reserve(foundCount);
            for (uint32_t i = 0; i < foundCount; i++) {
                uint32_t* itemPtr = hostOutput_ + (i * ITEM_SIZE32 + 1);
                Item item;
                item.threadId = itemPtr[0];
                const int16_t* ptr = reinterpret_cast<int16_t*>(&itemPtr[1]);
                item.endomorphism = ptr[0] & 0x7FFF;
                item.mode = (ptr[0] & 0x8000) != 0;
                item.increment = ptr[1];
                item.hash = reinterpret_cast<uint8_t*>(itemPtr + 2);
                foundItems.push_back(item);
            }
        }

        return CallKernel();
    }

    bool Engine::CheckHash(const uint8_t* hash, std::vector<Item>& foundItems,
                         int threadId, int increment, int endomorphism, int* okCount) {
        auto it = std::find_if(foundItems.begin(), foundItems.end(),
            [hash](const Item& item) {
                return ripemd160_comp_hash(item.hash, hash);
            });

        if (it != foundItems.end()) {
            foundItems.erase(it);
            (*okCount)++;
            return true;
        }

        printf("Expected item not found %s (thread=%d, incr=%d, endo=%d)\n",
               toHex(hash, 20).c_str(), threadId, increment, endomorphism);
        return false;
    }

    bool Engine::Check(Secp256K1* secp) {
        if (!initialized_) {
            return false;
        }

        printf("GPU: %s\n", deviceName_.c_str());

#ifdef FULLCHECK
        // Verify endianness
        get_endianness<<<1, 1>>>(deviceOutput_);
        CheckCudaError(hipGetLastError(), "Endianness check kernel");
        CheckCudaError(hipMemcpy(hostOutput_, deviceOutput_, 1, hipMemcpyDeviceToHost),
                      "Copy endianness result");
        isLittleEndian_ = *hostOutput_ != 0;
        printf("Endianness: %s\n", (isLittleEndian_ ? "Little" : "Big"));

        // Verify modular multiplication
        Int a, b, r, c;
        a.Rand(256);
        b.Rand(256);
        c.ModMulK1(&a, &b);
        
        std::memcpy(hostKeys_, a.bits64, BIFULLSIZE);
        std::memcpy(hostKeys_ + 5, b.bits64, BIFULLSIZE);
        CheckCudaError(hipMemcpy(deviceKeys_, hostKeys_, BIFULLSIZE * 2, hipMemcpyHostToDevice),
                      "Copy values for multiplication check");

        chekc_mult<<<1, 1>>>(deviceKeys_, deviceKeys_ + 5, (uint64_t*)deviceOutput_);
        CheckCudaError(hipGetLastError(), "Multiplication check kernel");
        CheckCudaError(hipMemcpy(hostOutput_, deviceOutput_, BIFULLSIZE, hipMemcpyDeviceToHost),
                      "Copy multiplication result");
        std::memcpy(r.bits64, hostOutput_, BIFULLSIZE);

        if (!c.IsEqual(&r)) {
            printf("\nModular Mult wrong:\nR=%s\nC=%s\n",
                   toHex((uint8_t*)r.bits64, BIFULLSIZE).c_str(),
                   toHex((uint8_t*)c.bits64, BIFULLSIZE).c_str());
            return false;
        }

        // Verify hash computation
        uint8_t h[20], hc[20];
        Point pi;
        pi.x.Rand(256);
        pi.y.Rand(256);
        secp->GetHash160(pi, false, h);
        secp->GetHash160(pi, true, hc);

        std::memcpy(hostKeys_, pi.x.bits64, BIFULLSIZE);
        std::memcpy(hostKeys_ + 5, pi.y.bits64, BIFULLSIZE);
        CheckCudaError(hipMemcpy(deviceKeys_, hostKeys_, BIFULLSIZE * 2, hipMemcpyHostToDevice),
                      "Copy values for hash check");

        chekc_hash160<<<1, 1>>>(deviceKeys_, deviceKeys_ + 5, deviceOutput_);
        CheckCudaError(hipGetLastError(), "Hash check kernel");
        CheckCudaError(hipMemcpy(hostOutput_, deviceOutput_, 64, hipMemcpyDeviceToHost),
                      "Copy hash results");

        if (!ripemd160_comp_hash((uint8_t*)hostOutput_, h)) {
            printf("\nGetHash160 wrong:\n%s\n%s\n",
                   toHex((uint8_t*)hostOutput_, 20).c_str(),
                   toHex(h, 20).c_str());
            return false;
        }

        if (!ripemd160_comp_hash((uint8_t*)(hostOutput_ + 5), hc)) {
            printf("\nGetHash160Comp wrong:\n%s\n%s\n",
                   toHex((uint8_t*)(hostOutput_ + 5), 20).c_str(),
                   toHex(h, 20).c_str());
            return false;
        }
#endif // FULLCHECK

        std::vector<Point> points(totalThreads_);
        std::vector<Point> points2(totalThreads_);
        Int k;

        if (searchMode_ == SearchMode::Both) {
            printf("Warning, Check function does not support BOTH_MODE, use either compressed or uncompressed");
            return true;
        }

        const bool searchComp = (searchMode_ == SearchMode::Compressed);
        const uint32_t seed = static_cast<uint32_t>(time(nullptr));
        printf("Seed: %u\n", seed);
        rseed(seed);

        int nbOK[6] = {0};
        int nbFoundCPU[6] = {0};
        std::vector<Item> foundItems;

        // Initialize points
        for (int i = 0; i < totalThreads_; i++) {
            k.Rand(256);
            points[i] = secp->ComputePublicKey(&k);
            k.Add((uint64_t)GRP_SIZE / 2);
            points2[i] = secp->ComputePublicKey(&k);
        }

        // Set test prefixes and keys
        SetPrefixes({0xFEFE, 0x1234});
        SetKeys(points2);
        
        const double t0 = Timer::get_tick();
        Launch(foundItems, true);
        const double t1 = Timer::get_tick();
        Timer::printResult("Key", 6 * STEP_SIZE * totalThreads_, t0, t1);

        printf("ComputeKeys() found %zu items, CPU check...\n", foundItems.size());

        // Prepare endomorphism constants
        Int beta, beta2;
        beta.SetBase16("7ae96a2b657c07106e64479eac3434e99cf0497512f58995c1396c28719501ee");
        beta2.SetBase16("851695d49a83f8ef919bb86153cbcb16630fb68aed0a766a3ec693d68e6afa40");

        // Verify results
        bool ok = true;
        for (int j = 0; j < totalThreads_; j++) {
            for (int i = 0; i < STEP_SIZE; i++) {
                Point pt = points[j];
                Point p1 = points[j];
                Point p2 = points[j];
                p1.x.ModMulK1(&beta);
                p2.x.ModMulK1(&beta2);
                points[j] = secp->NextKey(points[j]);

                uint8_t h[20];
                auto checkPoint = [&](const Point& p, int endo, int incr, int idx) {
                    secp->GetHash160(SearchType::P2PKH, searchComp, p, h);
                    const PrefixT pr = *(PrefixT*)h;
                    if (pr == 0xFEFE || pr == 0x1234) {
                        nbFoundCPU[idx]++;
                        ok &= CheckHash(h, foundItems, j, incr, endo, nbOK + idx);
                    }
                };

                // Check all variations
                checkPoint(pt, 0, i, 0);
                checkPoint(p1, 1, i, 1);
                checkPoint(p2, 2, i, 2);

                // Check symmetric versions
                pt.y.ModNeg();
                p1.y.ModNeg();
                p2.y.ModNeg();

                checkPoint(pt, 0, -i, 3);
                checkPoint(p1, 1, -i, 4);
                checkPoint(p2, 2, -i, 5);
            }
        }

        if (ok && !foundItems.empty()) {
            ok = false;
            printf("Unexpected item found!\n");
        }

        if (!ok) {
            const int totalFound = std::accumulate(nbFoundCPU, nbFoundCPU + 6, 0);
            printf("CPU found %d items\n", totalFound);

            printf("GPU: point   correct [%d/%d]\n", nbOK[0], nbFoundCPU[0]);
            printf("GPU: endo #1 correct [%d/%d]\n", nbOK[1], nbFoundCPU[1]);
            printf("GPU: endo #2 correct [%d/%d]\n", nbOK[2], nbFoundCPU[2]);
            printf("GPU: sym/point   correct [%d/%d]\n", nbOK[3], nbFoundCPU[3]);
            printf("GPU: sym/endo #1 correct [%d/%d]\n", nbOK[4], nbFoundCPU[4]);
            printf("GPU: sym/endo #2 correct [%d/%d]\n", nbOK[5], nbFoundCPU[5]);
            printf("GPU/CPU check Failed!\n");
        } else {
            printf("GPU/CPU check OK\n");
        }

        return ok;
    }

    void Engine::PrintCudaInfo() {
        const char* computeModes[] = {
            "Multiple host threads",
            "Only one host thread",
            "No host thread",
            "Multiple process threads",
            "Unknown",
            nullptr
        };

        int deviceCount = 0;
        hipError_t err = hipGetDeviceCount(&deviceCount);
        if (err != hipSuccess) {
            printf("GPUEngine: CudaGetDeviceCount %s\n", hipGetErrorString(err));
            return;
        }

        if (deviceCount == 0) {
            printf("GPUEngine: No available CUDA devices\n");
            return;
        }

        for (int i = 0; i < deviceCount; i++) {
            err = hipSetDevice(i);
            if (err != hipSuccess) {
                printf("GPUEngine: hipSetDevice(%d) %s\n", i, hipGetErrorString(err));
                continue;
            }

            hipDeviceProp_t props;
            hipGetDeviceProperties(&props, i);
            printf("GPU #%d %s (%dx%d cores) (Cap %d.%d) (%.1f MB) (%s)\n",
                   i, props.name, props.multiProcessorCount,
                   ConvertSMVerToCores(props.major, props.minor),
                   props.major, props.minor,
                   props.totalGlobalMem / 1048576.0,
                   computeModes[props.computeMode]);
        }
    }

    int Engine::GetThreadCount() const {
        return totalThreads_;
    }

    int Engine::GetGroupSize() {
        return GRP_SIZE;
    }

    void Engine::SetSearchMode(SearchMode mode) {
        searchMode_ = mode;
    }

    void Engine::SetSearchType(SearchType type) {
        searchType_ = type;
    }

    std::string Engine::GetDeviceName() const {
        return deviceName_;
    }

    void Engine::GenerateCode(Secp256K1* secp, int size) {
        // Implementation depends on specific requirements
        throw std::runtime_error("GenerateCode not implemented in this version");
    }
} // namespace GPUEngine
